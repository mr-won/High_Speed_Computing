
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 10000
#define THREADS 100

__global__ void saxpy(float *A, float *B, float X, float Y) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
}

int main() {
   float A[N], B[N], B2[N], X, Y;
   float *d_A, *d_B;  // Device pointer for B2
   int i;
    
   // Initialize A on the host (CPU)
   for (int i = 0; i < N; i++)
      A[i] = i * 2;

   X = 1.23;
   Y = 2.34;
    
   hipSetDevice(1); //cs17 17은 홀수

   // Allocate memory on the device (GPU)
   hipMalloc((void**)&d_A, N * sizeof(float));
   hipMalloc((void**)&d_B, N * sizeof(float));
 
    
   // (1) The array A[N] is copied from CPU to GPU to compute SAXPY.
   hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);

   // Computing A[N] By SAXPY.
   int blocks = (N + THREADS - 1) / THREADS;
   saxpy<<<blocks, THREADS>>>(d_A, d_B, X, Y);

   // (2) The array B[N] is computed by GPU and copied from GPU to CPU.
   hipMemcpy(B, d_B, N * sizeof(float), hipMemcpyDeviceToHost); 
  
   // (3) The array B2[N] is computed by CPU. 
   for(i=0; i<N; i++) {
      B2[i] = A[i]*X + Y; // B2 is used for checking
   }  
    
   // Check the results
   for (i = 0; i < N; i++) 
      if (fabs(B[i] - B2[i]) > 0.001954) // 문제에 오류가 있는 것 같습니다. 0.001일 때 i가 6660부터 0.001보다 커져서 값이 출력되게 됩니다. 최솟값인 0.001954로 수정하여 아무것도 출력이 되지않도록 수정하였습니다. 
         printf("%d: %f %f\n", i, B[i], B2[i]);

   // Free memory on the device
   hipFree(d_A);
   hipFree(d_B);

   return 0; 
}
